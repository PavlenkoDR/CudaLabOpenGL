#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "kernel.h"

#include <stdio.h>

__global__ void MoveParticlesKernel(Particle* particles, Border* border)
{
	//particles[threadIdx.x].Move(*border);
	auto sumDistance = 0.;
	while (sumDistance < particles[threadIdx.x].speed)
	{
		auto localDistance = particles[threadIdx.x].speed - sumDistance;
		auto cosA = particles[threadIdx.x].direction.x / sqrt(pow(particles[threadIdx.x].direction.x, 2) + pow(particles[threadIdx.x].direction.y, 2) + pow(particles[threadIdx.x].direction.z, 2));
		auto cosB = particles[threadIdx.x].direction.y / sqrt(pow(particles[threadIdx.x].direction.x, 2) + pow(particles[threadIdx.x].direction.y, 2) + pow(particles[threadIdx.x].direction.z, 2));
		auto hypotenuseXY = localDistance * cosB;
		auto deltaZ = particles[threadIdx.x].direction.z * hypotenuseXY * cosA / abs(particles[threadIdx.x].direction.z);
		auto deltaY = particles[threadIdx.x].direction.y * sqrt(pow(hypotenuseXY, 2) - pow(deltaZ, 2)) / abs(particles[threadIdx.x].direction.y);
		auto deltaX = particles[threadIdx.x].direction.x * sqrt(pow(localDistance, 2) - pow(hypotenuseXY, 2)) / abs(particles[threadIdx.x].direction.x);

		auto fromDistance = sqrt(pow(deltaX, 2) + pow(deltaY, 2) + pow(deltaZ, 2));
		/*std::cout << "deltaX   " << deltaX << std::endl;
		std::cout << "deltaY   " << deltaY << std::endl;
		std::cout << "deltaZ   " << deltaZ << std::endl;
		std::cout << "distance " << fromDistance << std::endl;*/

		if ((border->first.x > particles[threadIdx.x].position.x - particles[threadIdx.x].radius + deltaX) && (border->first.x - (particles[threadIdx.x].position.x - particles[threadIdx.x].radius) > 0.))
		{
			particles[threadIdx.x].direction.x = -particles[threadIdx.x].direction.x;
			auto oldDeltaX = deltaX;
			deltaX = border->first.x - particles[threadIdx.x].position.x + particles[threadIdx.x].radius;
			deltaY *= deltaX / oldDeltaX;
			deltaZ *= deltaX / oldDeltaX;
		}
		if ((particles[threadIdx.x].position.x + deltaX + particles[threadIdx.x].radius > border->second.x) && (border->second.x - (particles[threadIdx.x].position.x + particles[threadIdx.x].radius) > 0.))
		{
			particles[threadIdx.x].direction.x = -particles[threadIdx.x].direction.x;
			auto oldDeltaX = deltaX;
			deltaX = border->second.x - particles[threadIdx.x].position.x - particles[threadIdx.x].radius;
			deltaY *= deltaX / oldDeltaX;
			deltaZ *= deltaX / oldDeltaX;
		}
		if ((border->first.y > particles[threadIdx.x].position.y - particles[threadIdx.x].radius + deltaY) && (border->first.y - (particles[threadIdx.x].position.y - particles[threadIdx.x].radius) > 0.))
		{
			particles[threadIdx.x].direction.y = -particles[threadIdx.x].direction.y;
			auto oldDeltaY = deltaY;
			deltaY = border->first.y - particles[threadIdx.x].position.y + particles[threadIdx.x].radius;
			deltaX *= deltaY / oldDeltaY;
			deltaZ *= deltaY / oldDeltaY;
		}
		if ((particles[threadIdx.x].position.y + deltaY + particles[threadIdx.x].radius > border->second.y) && (border->second.y - (particles[threadIdx.x].position.y + particles[threadIdx.x].radius) > 0.))
		{
			particles[threadIdx.x].direction.y = -particles[threadIdx.x].direction.y;
			auto oldDeltaY = deltaY;
			deltaY = border->second.y - particles[threadIdx.x].position.y - particles[threadIdx.x].radius;
			deltaX *= deltaY / oldDeltaY;
			deltaZ *= deltaY / oldDeltaY;
		}
		if ((border->first.z > particles[threadIdx.x].position.z - particles[threadIdx.x].radius + deltaZ) && (border->first.z - (particles[threadIdx.x].position.z - particles[threadIdx.x].radius) > 0.))
		{
			particles[threadIdx.x].direction.z = -particles[threadIdx.x].direction.z;
			auto oldDeltaZ = deltaZ;
			deltaZ = border->first.z - particles[threadIdx.x].position.z + particles[threadIdx.x].radius;
			deltaX *= deltaZ / oldDeltaZ;
			deltaY *= deltaZ / oldDeltaZ;
		}
		if ((particles[threadIdx.x].position.z + deltaZ + particles[threadIdx.x].radius > border->third.z) && (border->third.z - (particles[threadIdx.x].position.z + particles[threadIdx.x].radius) > 0.))
		{
			particles[threadIdx.x].direction.z = -particles[threadIdx.x].direction.z;
			auto oldDeltaZ = deltaZ;
			deltaZ = border->third.z - particles[threadIdx.x].position.z - particles[threadIdx.x].radius;
			deltaX *= deltaZ / oldDeltaZ;
			deltaY *= deltaZ / oldDeltaZ;
		}
		sumDistance += sqrt(pow(deltaX, 2) + pow(deltaY, 2) + pow(deltaZ, 2));
		/*std::cout << "new deltaX   " << deltaX << std::endl;
		std::cout << "new deltaY   " << deltaY << std::endl;
		std::cout << "new deltaZ   " << deltaZ << std::endl;
		std::cout << "new distance " << sumDistance << std::endl;
		std::cout << std::endl;*/
		particles[threadIdx.x].position.x += deltaX;
		particles[threadIdx.x].position.y += deltaY;
		particles[threadIdx.x].position.z += deltaZ;
	}
}

std::shared_ptr<KernelCUDA> KernelCUDAInstanse::instance = nullptr;

std::shared_ptr<KernelCUDA> KernelCUDAInstanse::getInstanse(std::vector<Particle>* particles, Border* border)
{
	if (!instance)
	{
		instance = std::make_shared<KernelCUDA>(particles, border);
	}
	return instance;
}

hipError_t KernelCUDA::_MoveParticlesCUDA()
{
	hipError_t cudaStatus;
	MoveParticlesKernel << <1, particles->size() >> > (particlesDevice, borderDevice);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "MoveParticlesKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}
	else
	{
		cudaStatus = hipDeviceSynchronize();
	}
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching MoveParticlesKernel!\n", cudaStatus);
	}
	else
	{
		cudaStatus = hipMemcpy(&(*particles)[0], particlesDevice, particles->size() * sizeof(Particle), hipMemcpyDeviceToHost);
	}

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}

	return cudaStatus;
}

int KernelCUDA::MoveParticlesCUDA()
{
	hipError_t cudaStatus = _MoveParticlesCUDA();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	return 0;
}

KernelCUDA::KernelCUDA(std::vector<Particle>* particles, Border* border) : particles(particles), border(border)
{
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
	else
	{
		cudaStatus = hipMalloc((void**)&particlesDevice, particles->size() * sizeof(Particle));
	}

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	else
	{
		cudaStatus = hipMalloc((void**)&borderDevice, sizeof(Border));
	}

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	else
	{
		cudaStatus = hipMemcpy(borderDevice, border, sizeof(Border), hipMemcpyHostToDevice);
	}

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}
	else
	{
		cudaStatus = hipMemcpy(particlesDevice, &(*particles)[0], particles->size() * sizeof(Particle), hipMemcpyHostToDevice);
	}

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}
}

KernelCUDA::~KernelCUDA()
{
	hipFree(borderDevice);
	hipFree(particlesDevice);
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
	}
}
